#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <iomanip>

#include "../../power.hpp"

#define THREADS	1024
#define BLOCKS	64
#define N 268435456
#define M 500000
//#define M 1048576

__global__ void kernel(int *x, int *y) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int r1, r2, r3;

	r1 = x[idx];
	r2 = x[idx+1];

	for (int i = 0; i < M; i++) {
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
	}

	y[idx] = r2;
	

}

using namespace std;

int main(void) {

	hipSetDevice(2);

	int *dev_x, *dev_y, *host;

	hipEvent_t t1, t2;
	float time;

	hipStream_t stream;
	hipStreamCreate(&stream);

	hipEventCreate(&t1);
	hipEventCreate(&t2);

	/* ---- long term power measurement ---- */
/*	Power pl(0, 0, true, 
		"/work/bzcdress/cuda/intadd_raw_power_long.dat", 
		"/work/bzcdress/cuda/intadd_raw_utilization_long.dat");
	pl.run();
*/
	hipHostAlloc((void**)&host, N * sizeof(int), hipHostMallocDefault);
	hipMalloc((void**)&dev_x, N * sizeof(int));
	hipMalloc((void**)&dev_y, N * sizeof(int));


	// Initialize
	memset(host, 0, N * sizeof(int));

	hipMemcpy(dev_x, host, N * sizeof(int), hipMemcpyHostToDevice);
	
	cout << "# ThreadNo BlockNo Block1P Block1U BlockNo Block2P Block2U ... BlockNo BlockNP BlockNU" << endl;

	cout << setprecision(7);

	for (int threads = 511; threads < 515; threads++) {
		cout << threads << " ";
		for (int blocks = 64; blocks < BLOCKS + 1; blocks++) {
			Power pk(0,2,false);
			cout << blocks << " " << flush;

			hipEventRecord(t1, 0);
			kernel<<< dim3(blocks), dim3(threads) >>>(dev_x, dev_y);

			sleep(1);
			pk.run();
			hipEventRecord(t2, 0);
			hipEventSynchronize(t2);

			pk.halt();
			hipEventElapsedTime(&time, t1, t2);
			
			cout << pk.getPowerMean() << " " << pk.getUtilizationMean() << " " << time - 1000.0 << " " << flush;
/*			pk.writeToFile(
				"/work/bzcdress/cuda/intadd_raw_power.dat",
				"/work/bzcdress/cuda/intadd_raw_utilization.dat",
				blocks, threads);*/
		}
		cout << endl;
	}

	hipMemcpy(host, dev_y, N * sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventDestroy(t1);
	hipEventDestroy(t2);

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(host);

	hipStreamDestroy(stream);

//	pl.halt();

	return 0;

}
