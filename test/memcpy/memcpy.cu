#include "hip/hip_runtime.h"
#include <iostream>

#include "../../power.hpp"

#define N 536870912
#define M 4

__global__ void kernel() {

}

using namespace std;

int main(void) {

	unsigned long long *dev, *host;

	hipEvent_t t1, t2;
	float time1, time2;

	hipEventCreate(&t1);
	hipEventCreate(&t2);

	hipHostAlloc((void**)&host, N * sizeof(unsigned long long), hipHostMallocDefault);
	hipMalloc((void**)&dev, N * sizeof(unsigned long long));

	Power p(0,0);

	// Make sure device is not in sleep mode
	hipMemcpy(dev, host, 1024 * sizeof(unsigned long long), hipMemcpyHostToDevice);
	
	// Start measurement
	p.run();
	sleep(1);

	hipEventRecord(t1, 0);
	for (int i = 0; i < M; i++) {
		hipMemcpy(dev, host, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
	}
	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);

	hipEventElapsedTime(&time1, t1, t2);

	sleep(1);

	hipEventRecord(t1, 0);
	for (int i = 0; i < M; i++) {
		hipMemcpy(host, dev, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	}
	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);

	hipEventElapsedTime(&time2, t1, t2);

	sleep(1);
	p.halt();

	hipEventDestroy(t1);
	hipEventDestroy(t2);

	hipFree(dev);
	hipFree(host);

	cin.get();

	//cout << time1 << " ms\t" << time2 << " ms" << endl;

	return 0;

}
