#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <stdexcept>
#include <iomanip>

#include "../../power.hpp"

#define VECDIM	16384
#define BLOCKS	64
#define THREADS	1024
#define ITER	1200

using namespace std;

__global__ void dotprod(float *vectors, float *dotprods) {

	int idx = blockDim.x * gridDim.x + threadIdx.x;

	for (int n = 0; n < ITER; n++) {
		
		float a = 0.0;
		
		for (int i = 0; i < VECDIM; i++) {
			a += pow(vectors[idx + i * VECDIM], 2.0f);
		}

		dotprods[idx] = sqrt(a);
	}

}

int main(void) {

	/* ---- Change device ---- */

	hipSetDevice(2);

	/* ---- Initialize everything ---- */

	hipEvent_t t1, t2;
	float time;

	hipEventCreate(&t1);
	hipEventCreate(&t2);

	/* ---- long term power measurement ---- */
/*	Power pl(0, 1, true, 
		"/work/bzcdress/cuda/vecmet_raw_power_long.dat", 
		"/work/bzcdress/cuda/vecmet_raw_utilization_long.dat");
	pl.run();
*/
	float *dev_vectors, *host_vectors;
	float *dev_dotprods, *host_dotprods;

	/* ---- Alloc vectors ---- */
	host_dotprods = new float[THREADS * BLOCKS];
	host_vectors = new float[THREADS * BLOCKS * VECDIM];
	for (int i = 0; i < THREADS * BLOCKS * VECDIM; i++) {
		host_vectors[i] = 1.0;
	}

	hipMalloc((void**) &dev_dotprods, BLOCKS * THREADS * sizeof(float));
	hipMalloc((void**) &dev_vectors, BLOCKS * THREADS * VECDIM * sizeof(float));

	/* ---- Copy vectors to device */

	hipMemcpy(dev_vectors, host_vectors, BLOCKS * THREADS * VECDIM * sizeof(float), hipMemcpyHostToDevice);

	/* ---- Kernel call ---- */
	
	cout << setprecision(7);

	for (int i = 0; i < 1000; i++) {

			cout << i << " ";

			int blocks = 32;
			int threads = 512;

//	for (int threads = 511; threads < 515; threads++) {
//		cout << threads << " ";
//		for (int blocks = 64; blocks < BLOCKS + 1; blocks++) {
			Power pk(0,2,false);
//			cout << blocks << " ";

			hipEventRecord(t1, 0);
			dotprod<<< blocks, threads >>>(dev_vectors, dev_dotprods);

			sleep(1);
			pk.run();
			hipEventRecord(t2, 0);
			hipEventSynchronize(t2);
			
			pk.halt();
			hipEventElapsedTime(&time, t1, t2);

//			cout << pk.getPowerMean() << " " << pk.getUtilizationMean() << " " << time - 1000.0 << " " << flush;
			pk.writeToFile(
				"/work/bzcdress/cuda/statistics_raw_power.dat",
				"/work/bzcdress/cuda/statistics_raw_utilization.dat",
				32, 512);
//		}
//		cout << endl;
//	}
	}
	cout << endl;

	/* ---- Backcopy ---- */

	hipMemcpy(host_dotprods, dev_dotprods, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(dev_vectors);
	hipFree(dev_dotprods);

	delete[] host_dotprods;
	delete[] host_vectors;

//	pl.halt();

	return 0;

}
