#include "hip/hip_runtime.h"
#include <iostream>

#include "../../power.hpp"

#define THREADS 512
#define BLOCKS	48

#define VSIZE	8192
#define N		BLOCKS * THREADS * VSIZE

__global__ void kernel(unsigned long long *x, unsigned long long *y, unsigned long long *z) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int vidx = idx * VSIZE;

	unsigned long long t = 0;

	for (int j = 0; j < 4; j++) {
	for (int i = 0; i < VSIZE; i++) {
			t += x[vidx + i] * y[vidx + i];
	}
	}
	

	z[idx] = t;
	

}

using namespace std;

int main(void) {

	unsigned long long *dev_x, *dev_y, *dev_z, *host;

	hipEvent_t t1, t2;
	float time;

	hipStream_t stream;
	hipStreamCreate(&stream);

	hipEventCreate(&t1);
	hipEventCreate(&t2);

	hipHostAlloc((void**)&host, N * sizeof(unsigned long long), hipHostMallocDefault);
	hipMalloc((void**)&dev_x, N * sizeof(unsigned long long));
	hipMalloc((void**)&dev_y, N * sizeof(unsigned long long));
	hipMalloc((void**)&dev_z, THREADS * BLOCKS * sizeof(unsigned long long));

	Power p(0,0);

	// Initialize
	memset(host, 0, N * sizeof(unsigned long long));

	// Start power measurement
	p.run();
	sleep(1);

	hipMemcpy(dev_x, host, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, host, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
	hipMemcpy(dev_z, host, THREADS * BLOCKS * sizeof(unsigned long long), hipMemcpyHostToDevice);
	
	sleep(1);
	
	hipEventRecord(t1, 0);
	
	kernel<<< dim3(BLOCKS), dim3(THREADS), 0, stream >>>(dev_x, dev_y, dev_z);

	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);
	hipEventElapsedTime(&time, t1, t2);

	sleep(1);

	hipMemcpy(host, dev_y, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	
	sleep(1);
	p.halt();

	hipEventDestroy(t1);
	hipEventDestroy(t2);

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	hipFree(host);

	hipStreamDestroy(stream);

	cin.get();

	cout << time << " ms" << endl;

	return 0;

}
