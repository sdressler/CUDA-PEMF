#include "hip/hip_runtime.h"
#include <iostream>

#include "../../power.hpp"

#define THREADS 512
#define BLOCKS	48

#define VSIZE	8192
#define N		BLOCKS * THREADS * VSIZE

__global__ void kernel(unsigned long long *x, unsigned long long *y, unsigned long long *z) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x * VSIZE;

	for (int i = 0; i < VSIZE; i++) {
		int _i = idx + i;
		z[_i] = x[_i] + y[_i];
	}

}

using namespace std;

int main(void) {

	unsigned long long *dev_x, *dev_y, *dev_z, *host;

	hipEvent_t t1, t2;
	float time;

	hipStream_t stream;
	hipStreamCreate(&stream);

	hipEventCreate(&t1);
	hipEventCreate(&t2);

	hipHostAlloc((void**)&host, N * sizeof(unsigned long long), hipHostMallocDefault);
	hipMalloc((void**)&dev_x, N * sizeof(unsigned long long));
	hipMalloc((void**)&dev_y, N * sizeof(unsigned long long));
	hipMalloc((void**)&dev_z, N * sizeof(unsigned long long));

	Power p(0,0);

	// Initialize
	memset(host, 0, N * sizeof(unsigned long long));

	// Start power measurement
	p.run();
	sleep(1);

	hipMemcpy(dev_x, host, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
	
	sleep(1);
	
	hipEventRecord(t1, 0);
	
	kernel<<< dim3(BLOCKS), dim3(THREADS), 0, stream >>>(dev_x, dev_y, dev_z);

	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);
	hipEventElapsedTime(&time, t1, t2);

	sleep(1);

	hipMemcpy(host, dev_y, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	
	sleep(1);
	p.halt();

	hipEventDestroy(t1);
	hipEventDestroy(t2);

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	hipFree(host);

	hipStreamDestroy(stream);

	cin.get();

	cout << time << " ms" << endl;

	return 0;

}
