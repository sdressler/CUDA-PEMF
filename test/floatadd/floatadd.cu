#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>

#include "../../power.hpp"

#define THREADS	1024
#define BLOCKS	64
#define N THREADS * BLOCKS
//#define N 268435456
//#define M 5000000
//#define M 1048576
#define M 500000

__global__ void kernel(float *x, float *y) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float r1, r2, r3;

	r1 = x[idx];
	r2 = x[idx+1];

	for (int i = 0; i < M; i++) {
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
			r3 = r1 + r2;
			r2 = r3 + r1;
			r1 = r2 + r3;
	}

	y[idx] = r2;
	

}

using namespace std;

int main(void) {

	hipSetDevice(0);

	float *dev_x, *dev_y, *host;

	hipEvent_t t1, t2;
	float time;

	hipStream_t stream;
	hipStreamCreate(&stream);

	hipEventCreate(&t1);
	hipEventCreate(&t2);

	hipHostAlloc((void**)&host, N * sizeof(float), hipHostMallocDefault);
	hipMalloc((void**)&dev_x, N * sizeof(float));
	hipMalloc((void**)&dev_y, N * sizeof(float));


	// Initialize
	memset(host, 0, N * sizeof(float));


	hipMemcpy(dev_x, host, N * sizeof(float), hipMemcpyHostToDevice);
	
//	for (int blocks = 1; blocks < BLOCKS + 1; blocks++) {
//		int iblocks = 2;
//		for (int threads = 1; threads < THREADS + 1; threads++) {

	cout << "# ThreadNo BlockNo Block1P Block1U BlockNo Block2P Block2U ... BlockNo BlockNP BlockNU" << endl;

//	for (int threads = 2; threads < THREADS + 1; threads = 2) {
		
//	int threads = 304;
//	int blocks = 17;

		// Start power measurement
		//p.run();
	for (int threads = 480; threads < 577; threads += 32) {
		cout << threads << " " << flush;
		for (int blocks = 64; blocks < 65; blocks++) {
//			Power p(0,2,false);
//			sleep(1);
//			cout << threads*blocks << " " << flush;
			cout << blocks << " " << flush;

			hipEventRecord(t1, 0);
			kernel<<< dim3(blocks), dim3(threads) >>>(dev_x, dev_y);

//			sleep(1);
//			p.run();
			hipEventRecord(t2, 0);
			
			hipEventSynchronize(t2);
//			p.halt();
			hipEventElapsedTime(&time, t1, t2);
//			p.getMean();

//			sleep(5);

//			p.halt();
//			cout << p.getPowerMean() << " " << p.getUtilizationMean() << " " << time - 1000.0 << " " << flush;
			cout << time << flush;
//			p.writeToFile("raw_power.dat", "raw_utilization.dat", blocks, threads);
		}
		cout << endl;
	}

	hipMemcpy(host, dev_y, N * sizeof(float), hipMemcpyDeviceToHost);
/*	
	vector<float> power = p.getPower();
	for (vector<float>::iterator i = power.begin(); i != power.end(); ++i) {
		cout << *i << " ";
	}
	cout << endl;

	vector<int> u = p.getUtilization();
	for (vector<int>::iterator i = u.begin(); i != u.end(); ++i) {
		cout << *i << " ";
	}
	cout << endl;
*/
	hipEventDestroy(t1);
	hipEventDestroy(t2);

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(host);

	hipStreamDestroy(stream);

	return 0;

}
