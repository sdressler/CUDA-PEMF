#include "hip/hip_runtime.h"
#include <iostream>

#include "../../power.hpp"

#define THREADS 512
#define N 268435456
#define M 65536

__global__ void kernel(unsigned long long *x, unsigned long long *y) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j = 0; j < 16; j++) {
	for (int i = 0; i < M; i++) {
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
			y[idx] = x[idx];
	}
	}

}

using namespace std;

int main(void) {

	unsigned long long *dev_x, *dev_y, *host;

	hipEvent_t t1, t2;
	float time;

	hipStream_t stream;
	hipStreamCreate(&stream);

	hipEventCreate(&t1);
	hipEventCreate(&t2);

	hipHostAlloc((void**)&host, N * sizeof(unsigned long long), hipHostMallocDefault);
	hipMalloc((void**)&dev_x, N * sizeof(unsigned long long));
	hipMalloc((void**)&dev_y, N * sizeof(unsigned long long));

	Power p(0,0);

	// Initialize
	memset(host, 0, N * sizeof(unsigned long long));

	// Start power measurement
	p.run();
	sleep(1);

	hipMemcpy(dev_x, host, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
	
	sleep(1);
	
	hipEventRecord(t1, 0);
	
	kernel<<< dim3(48), dim3(THREADS), 0, stream >>>(dev_x, dev_y);

	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);
	hipEventElapsedTime(&time, t1, t2);

	sleep(1);

	hipMemcpy(host, dev_y, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	
	sleep(1);
	p.halt();

	hipEventDestroy(t1);
	hipEventDestroy(t2);

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(host);

	hipStreamDestroy(stream);

	cin.get();

	cout << time << " ms" << endl;

	return 0;

}
